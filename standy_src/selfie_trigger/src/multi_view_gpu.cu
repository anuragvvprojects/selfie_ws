#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <cstdio>
#include <iostream>
#include <cmath>
#include <algorithm>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <iostream>
#include <algorithm>
#include <limits>
using namespace cv;
using namespace std;

static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
	if(err!=hipSuccess)
	{
		fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)

texture<float, 2, hipReadModeElementType> tex_Im;
texture<float, 2, hipReadModeElementType> tex_Ir;


__global__ void costVolume( float *dIr, float *dCost, float *dCounter, float* dDepth, float* dMinm,bool dLast, float *dConf,float *dest ,float *P,float *B,int height,int width, int maxOrd, int nDepth)
{
  	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
  	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = yIndex*width+xIndex;
	float qmin = 0.1, qmax = 10.001;

	float maxRange=0.0;

	float step = qmax/(nDepth) ;
	dConf[idx]=0.0;
	dest[idx]=0.0;
	
	int order = 1;

	float dSum = 0.0;
	float dH[9];
	float dMax = 0.0;
	float dcostMin = 100000.0;
	float dEst = 1000.0;
	int countFail = 0;
	//dCounter[ 0 ] = dCounter[ 0 ] + 1;
	for(int di=0; di< nDepth ; di++)     //depth
	{
                  
    		float dval = qmin + di*step;
									//if(xIndex==100 && yIndex==100)	printf("%e\n",dval);
    	
    		for(int i = 0; i<9; i++)
			dH[i] = P[i] + dval * B[i];
									//if(xIndex==100 && yIndex==100){ for(int i = 0; i<9; i++) printf("%e ",dH[i]); printf("\n");} 
		float val = 0.0;
		//int patch_sz = (2*order + 1)*(2*order + 1);
		//int fail = 0;
		int countPass = 0;
		for(int patch_x = -1*order; patch_x <= order; patch_x++)
		{
			for(int patch_y = -1*order; patch_y <= order; patch_y++)
			{
				int nxIndex = xIndex + patch_x;
				int nyIndex = yIndex + patch_y;
				float u = dH[0] * (float)nyIndex + dH[1] * (float)nxIndex + dH[2];
				float v = dH[3] * (float)nyIndex + dH[4] * (float)nxIndex + dH[5];
				float w = dH[6] * (float)nyIndex + dH[7] * (float)nxIndex + dH[8];
				float neiImgCost = 0;
   				if(w>0 || w <0)
    				{
    					float neiIdx_y = u/w,neiIdx_x = v/w;
      					if(neiIdx_y > height || neiIdx_x > width || neiIdx_y < 0 || neiIdx_x < 0)
      						neiImgCost = 0;
      					else{
        					neiImgCost = abs(tex2D(tex_Im, neiIdx_x, neiIdx_y) - dIr[yIndex*width + xIndex]);//Shouldnt the second I value come from Ir rather than Im??
        					//fail = 255;
        					countPass++;
        				}
    				}
    	    
				val += neiImgCost;    //L1  
		    	}
		}


		if (countPass > 6 )
		{
			dCounter[ di*height*width + yIndex*width + xIndex ] ++;
		}
		dCost[ di*height*width + yIndex*width + xIndex ]+=val;
		


		dMax = max(dMax , val);
		dSum = dSum + val;
	}
	
	//delete[] dH;
}

__global__ void cost_refinement( float *dCost, float *dCounter, float* dDepth, float* dMinm, int height,int width, int maxOrd, int nDepth)
{
  	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
  	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = yIndex*width+xIndex;
	float qmin = 0.1, qmax = 10.001;
	float step = qmax/(nDepth) ;
	dMinm[yIndex*width + xIndex] = 1000.0f;
	dDepth[yIndex*width + xIndex] = qmin;
	for(int di=0; di< nDepth ; di++)     //depth
	{

		if((dCounter[ di*height*width + yIndex*width + xIndex ]  > 0))
		{
			float min_cost = dMinm[yIndex*width + xIndex];
			float cur_cost = dCost[ di*height*width + yIndex*width + xIndex ]/(float)dCounter[ di*height*width + yIndex*width + xIndex ];
			if (min_cost > cur_cost)
				{
					dMinm[yIndex*width + xIndex] = cur_cost;
					dDepth[yIndex*width + xIndex] = qmin + di*step;
				}
		}
		else
			dCost[ di*height*width + yIndex*width + xIndex ] = 100.0;

	}
}

void function(Mat& Ir,vector<Mat>& Im, vector<Mat>& R,vector<Mat>& T,Mat& Nv,Mat& K,Mat& Ki,Mat& confMat, Mat &borderFlag, Mat &IrEstDepth, int height, int width,int order,float dMin,float stepSz, int nDepth)
{
	if( !( Ir.rows == height ) || !( Ir.cols == width ) )
	{
		cout << " Input image is not of  same dimensions as heignt and width. Return;. " << endl; return;
	}

bool DEBUG = 0;
	Mat qmat=Mat::zeros(height,width,CV_32FC1);
	float *dIr;
	float *dIm;
	float *dCost;
	float *dCounter;	
	float *dDepth;
	float *dMinm;
	float *dest;
	float *dP;
	float *dB;
	float *dConf;
	//int *dBorderFlag;
	int maxOrd=order;
	const int tPx = height*width;
	const int imgSz = Ir.cols*Ir.rows*sizeof(float);
													if(DEBUG)cout << "1. -------------------------------------\nIr rows :" << Ir.rows << "Ir cols :"  << 		Ir.cols << "Ir type :"<< Ir.type()<<endl;
	hipMalloc((void **)&dIr, imgSz);								if(DEBUG)printf("hipMalloc dIr:\t%s\n", hipGetErrorString(hipGetLastError()));
													if(DEBUG)cout << "2. Copying dIr" << endl;
	SAFE_CALL(hipMemcpy(dIr,Ir.ptr(),imgSz,hipMemcpyHostToDevice),"CUDA Memcpy Host To Device Failed");//
	hipMalloc((void **)&dCost, nDepth*tPx*sizeof(float));						if(DEBUG)printf("3. hipMalloc dCost:\t%s\n", hipGetErrorString(hipGetLastError()));
	SAFE_CALL(hipMemset(dCost, 0, nDepth*tPx*sizeof(float)),"CUDA Memset Failed");			if(DEBUG)cout<<"dCost[100] : " << dCost[100] << endl;
	hipMalloc((void **)&dCounter, nDepth*tPx*sizeof(float));					if(DEBUG)printf("4. hipMalloc dCounter:\t%s\n", hipGetErrorString(hipGetLastError()));
	hipMemset(dCounter, 0, nDepth*tPx*sizeof(int));
	hipMalloc((void **)&dDepth, imgSz);								if(DEBUG)printf("5. hipMalloc dDepth:\t%s\n", hipGetErrorString(hipGetLastError()));
	hipMemset(dDepth, 0, tPx*sizeof(float));
	hipMalloc((void **)&dMinm, tPx*sizeof(float));							if(DEBUG)printf("6. hipMalloc dMinm:\t%s\n", hipGetErrorString(hipGetLastError()));
	hipMemset(dMinm, std::numeric_limits<float>::infinity(), tPx*sizeof(float));//cout << " Value of dMinm :" << dMinm[0] <<endl;
													if(DEBUG)cout<<"6.0 Size of Im :" << Im.size()<<endl;		

	dim3 blSz(8,32,1);								if(DEBUG)printf("17.dim3 blSz(8,32,1)  :\t%s\n", hipGetErrorString(hipGetLastError()));
	dim3 thSz(nDepth/blSz.x, (tPx)/blSz.y, 1);
	dim3 blockSize(16, 16, 1);
	dim3 gridSize((Ir.cols)/blockSize.x, (Ir.rows)/blockSize.y, 1);	

	hipMalloc((void **)&dIm, imgSz);						if(DEBUG)printf("8. hipMalloc dIm:\t%s\n", hipGetErrorString(hipGetLastError()));
	hipMalloc((void **)&dConf, tPx*sizeof(float));					if(DEBUG)printf("9. hipMalloc dConf:\t%s\n", hipGetErrorString(hipGetLastError()));
	hipMalloc((void **)&dest, tPx*sizeof(float));					if(DEBUG)printf("10.hipMalloc dest:\t%s\n", hipGetErrorString(hipGetLastError()));
	hipMalloc((void **)&dP, 3*3*sizeof(float));					if(DEBUG)printf("11.hipMalloc dP:\t%s\n", hipGetErrorString(hipGetLastError()));
	hipMalloc((void **)&dB, 3*3*sizeof(float));					if(DEBUG)printf("12.hipMalloc dB:\t%s\n", hipGetErrorString(hipGetLastError()));
													if(DEBUG)printf("13.Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));

	size_t pitch;
	float* d_texture; //Device texture
											if(DEBUG)cout << "24.Creating d_texture" << endl;
	hipMallocPitch((void**)&d_texture,&pitch, width * sizeof(float), height);
													if(DEBUG)printf("25.d_texture malloc:\t%s\n", hipGetErrorString(hipGetLastError()));	
//--------------FOR LOOP............................
	for(int i = 0 ; i < Im.size() ; i++ )
		{
													if(DEBUG)cout << "7.  image " << i << "-------------------------------------\n" << endl;
													if(DEBUG)cout << "14.Copying dIm" << endl;
			//SAFE_CALL(hipMemcpy(dIm,Im[i].ptr(),imgSz,hipMemcpyHostToDevice),"CUDA Memcpy Host To Device Failed");//
													if(DEBUG)printf("15.Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
													if(DEBUG)cout << "16.Copying done. Size of Im: "<< Im[i].size() << endl;

			Mat A = T[i]*Nv;//
			Mat P = K * R[i] * Ki;//
			Mat B = K * A * Ki;//
													if(DEBUG)cout << "19.dP\n" << P << "\n dB\n"<< B << "\n R\n"<< R[i] <<"\n T\n" << T[i] <<endl;
			hipMemcpy(dP, P.ptr(), 3*3*sizeof(float), hipMemcpyHostToDevice);
													if(DEBUG)printf("20.hipMemcpy(dP  :\t%s\n", hipGetErrorString(hipGetLastError()));
													if(DEBUG)cout << "21.dB" << endl;
			hipMemcpy(dB, B.ptr(), 3*3*sizeof(float), hipMemcpyHostToDevice);
													if(DEBUG)printf("22.hipMemcpy(dB  :\t%s\n", hipGetErrorString(hipGetLastError()));
													if(DEBUG)cout << "23.Copying done" << endl;
													if(DEBUG)cout << "26.Copying d_texture" << endl;
			hipMemcpy2D(d_texture, pitch, Im[i].ptr(), width * sizeof(float), width *sizeof(float), height, hipMemcpyHostToDevice); 
													if(DEBUG)printf("27.d_texture malloc Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
													if(DEBUG)cout << "28.bind_texture" << endl;
			hipBindTexture2D(NULL, tex_Im, d_texture, tex_Im.channelDesc, width, height, pitch) ;
													if(DEBUG)printf("29.hipBindTexture2D:\t%s\n", hipGetErrorString(hipGetLastError()));
													if(DEBUG)cout << "30.Copying done" << endl;
			//-------COstVolume Calculation------------------------------------------------------------------------------------------
			bool dLast = (i == (Im.size() -1));						if(DEBUG)cout << "31.Costvolume: dLast  "<<dLast<<"-----------------  :  " << dLast << endl;
			costVolume<<<gridSize,blockSize>>>(dIr, dCost, dCounter, dDepth, dMinm, dLast, dConf, dest, dP, dB, height, width , maxOrd, nDepth);
													if(DEBUG)printf("32.Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
													if(DEBUG)cout << "33.Costvolume done----------------------------------------------" << endl;
			//-----------------------------------------------------------------------------------------------------------------------
													
			double minE, maxE;
			cv::minMaxLoc(confMat, &minE, &maxE);						if(DEBUG)printf("34.Device Vasriable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
													if(DEBUG)cout<<"35.Confidence "<<minE<<' '<<maxE<<endl;
			hipUnbindTexture(tex_Im);							if(DEBUG)printf("36.Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
		}
		cost_refinement<<<gridSize,blockSize>>>(dCost, dCounter, dDepth, dMinm, height, width , maxOrd, nDepth);
		DEBUG = 1;
													//if(DEBUG)cout << "48.IrEstDepth type : " << IrEstDepth.type() << endl;

													if(DEBUG)cout<< "50. Testing Starts here----------------------------------------------" << endl;
		double minm, maxm;

		cv::Mat mDepth = cv::Mat::zeros(Ir.size(),IrEstDepth.type());
		hipMemcpy(mDepth.ptr(), dDepth, tPx*sizeof(float), hipMemcpyDeviceToHost);
		minMaxLoc(mDepth, &minm, &maxm);
													if(DEBUG)cout<<"51 :minmax of mDepth : "<<minm<<" , "<<maxm<<endl;
		cv::Mat mMinm = cv::Mat::zeros(Ir.size(),IrEstDepth.type());
		hipMemcpy(mMinm.ptr(), dMinm, tPx*sizeof(float), hipMemcpyDeviceToHost);
		minMaxLoc(mMinm, &minm, &maxm);
		/*normalize(mDepth,mDepth,0,1,NORM_MINMAX);						if(DEBUG)cout<<"52 :minmax of mMinm : "<<minm<<" , "<<maxm<<endl;
		imshow("dDepth",mDepth);
		waitKey();*/
		IrEstDepth = mDepth.clone();
		cv::Mat mCost[nDepth];
		cv::Mat mCounter[nDepth];								if(DEBUG)cout << " sIZE OF iM :" << Im.size()<< endl;
		hipFree(dIm);									if(DEBUG)printf("38.Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
		//hipFree(dConf);
												if(DEBUG)cout << "39.Memfree done" << endl;
		//hipMemcpy(confMat.ptr(), dConf, imgSz, hipMemcpyDeviceToHost);
												if(DEBUG)printf("40.Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
												if(DEBUG)cout << "41.Copying confMat done" << endl;
		//hipMemcpy(borderFlag.ptr(), dBorderFlag, height*width*sizeof(int), hipMemcpyDeviceToHost);
		//hipMemcpy(IrEstDepth.ptr(), dest, imgSz, hipMemcpyDeviceToHost);
												if(DEBUG)cout << "42.Copying IrEstDepth done" << endl;
												if(DEBUG)printf("43.Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
												//if(DEBUG)printf("%f\n",IrEstDepth.at<float>(100,100));
		hipFree(d_texture);								if(DEBUG)printf("37.Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));

	
		hipFree(dConf);								if(DEBUG)printf("44.Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
		hipFree(dest);									if(DEBUG)printf("45.Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
		hipFree(dP);									if(DEBUG)printf("46.Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
		hipFree(dB);									if(DEBUG)printf("47.Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
												//float* fCost = (float*)malloc(nDepth * tPx * sizeof(float));
/*													//hipMemcpy(fCost, dCost, nDepth*tPx*sizeof(float), hipMemcpyDeviceToHost);
		for(int i = 0; i < nDepth ; i++)
		{
			mCost[i] = cv::Mat::zeros(Ir.size(),IrEstDepth.type());
			hipMemcpy(mCost[i].ptr(), dCost+(i*tPx), tPx*sizeof(float), hipMemcpyDeviceToHost);
			//minMaxLoc(mCost[i], &minm, &maxm);
//													if(DEBUG)cout<<"53 :minmax of mCost["<<i<<"] : "<<minm<<" , "<<maxm<<endl;
											//if(DEBUG)cout<<"53.1 :mcost and fCost compare : " << mCost[i].at<float>(0,1) << " , " << *(fCost+(i*tPx)+1) << endl;

			Mat matTemp;
			normalize(mCost[i],matTemp,0,1,NORM_MINMAX);
			imshow("costVolume",matTemp);
			waitKey(30);

			mCounter[i] = cv::Mat::zeros(Ir.size(),IrEstDepth.type());
			hipMemcpy(mCounter[i].ptr(), dCounter+(i*tPx), tPx*sizeof(float), hipMemcpyDeviceToHost);
			//minMaxLoc(mCounter[i], &minm, &maxm);
//													if(DEBUG)cout<<"54 :minmax of mCounter["<<i<<"] : "<<minm<<" , "<<maxm<<endl;
		}
*/
	hipFree(dIr);//printf("Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
	hipFree(dCost);
	hipFree(dCounter);
	hipFree(dMinm);
	hipFree(dDepth);	
}



