#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <cstdio>
#include <iostream>
#include <cmath>
#include <algorithm>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/device_functions.h>
#include <iostream>
#include <algorithm>
#include <limits>
using namespace cv;
using namespace std;

static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
	if(err!=hipSuccess)
	{
		fprintf(stderr,"%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n",msg,file_name,line_number,hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}

#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)

texture<float, 2, hipReadModeElementType> tex_Im;
texture<float, 2, hipReadModeElementType> tex_Ir;


__global__ void costVolume( float *dIr, float *dCost, float *dCounter, float* dDepth, float* dMinm,bool dLast, float *dConf,float *dest ,float *P,float *B,int height,int width, int maxOrd, int nDepth)
{
  	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
  	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = yIndex*width+xIndex;
	float qmin = 0.1, qmax = 10.001;

	float maxRange=0.0;

	float step = qmax/(nDepth) ;
	dConf[idx]=0.0;
	dest[idx]=0.0;
	
	int order = 1;

	float dSum = 0.0;
	float dH[9];
	float dMax = 0.0;
	float dcostMin = 100000.0;
	float dEst = 1000.0;
	int countFail = 0;
	//dCounter[ 0 ] = dCounter[ 0 ] + 1;
	for(int di=0; di< nDepth ; di++)     //depth
	{
                  
    		float dval = qmin + di*step;
									//if(xIndex==100 && yIndex==100)	printf("%e\n",dval);
    	
    		for(int i = 0; i<9; i++)
			dH[i] = P[i] + dval * B[i];
									//if(xIndex==100 && yIndex==100){ for(int i = 0; i<9; i++) printf("%e ",dH[i]); printf("\n");} 
		float val = 0.0;
		//int patch_sz = (2*order + 1)*(2*order + 1);
		//int fail = 0;
		int countPass = 0;
		for(int patch_x = -1*order; patch_x <= order; patch_x++)
		{
			for(int patch_y = -1*order; patch_y <= order; patch_y++)
			{
				int nxIndex = xIndex + patch_x;
				int nyIndex = yIndex + patch_y;
				float u = dH[0] * (float)nyIndex + dH[1] * (float)nxIndex + dH[2];
				float v = dH[3] * (float)nyIndex + dH[4] * (float)nxIndex + dH[5];
				float w = dH[6] * (float)nyIndex + dH[7] * (float)nxIndex + dH[8];
				float neiImgCost = 0;
   				if(w>0 || w <0)
    				{
    					float neiIdx_y = u/w,neiIdx_x = v/w;
      					if(neiIdx_y > height || neiIdx_x > width || neiIdx_y < 0 || neiIdx_x < 0)
      						neiImgCost = 0;
      					else{
        					neiImgCost = abs(tex2D(tex_Im, neiIdx_x, neiIdx_y) - dIr[yIndex*width + xIndex]);//Shouldnt the second I value come from Ir rather than Im??
        					//fail = 255;
        					countPass++;
        				}
    				}
    	    
				val += neiImgCost;    //L1  
		    	}
		}


		if (countPass > 6 )
		{
			dCounter[ di*height*width + yIndex*width + xIndex ] ++;
		}
		dCost[ di*height*width + yIndex*width + xIndex ]+=val;
		


		dMax = max(dMax , val);
		dSum = dSum + val;
	}
	
	//delete[] dH;
}

__global__ void cost_refinement( float *dCost, float *dCounter, float* dDepth, float* dMinm, int height,int width, int maxOrd, int nDepth)
{
  	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
  	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = yIndex*width+xIndex;
	float qmin = 0.1, qmax = 10.001;
	float step = qmax/(nDepth) ;
	dMinm[yIndex*width + xIndex] = 1000.0f;
	dDepth[yIndex*width + xIndex] = qmin;
	for(int di=0; di< nDepth ; di++)     //depth
	{

		if((dCounter[ di*height*width + yIndex*width + xIndex ]  > 0))
		{
			float min_cost = dMinm[yIndex*width + xIndex];
			float cur_cost = dCost[ di*height*width + yIndex*width + xIndex ]/(float)dCounter[ di*height*width + yIndex*width + xIndex ];
			if (min_cost > cur_cost)
				{
					dMinm[yIndex*width + xIndex] = cur_cost;
					dDepth[yIndex*width + xIndex] = qmin + di*step;
				}
		}
		else
			dCost[ di*height*width + yIndex*width + xIndex ] = 100.0;

	}
}

void function(Mat& Ir,vector<Mat>& Im, vector<Mat>& R,vector<Mat>& T,Mat& Nv,Mat& K,Mat& Ki,Mat& confMat, Mat &borderFlag, Mat &IrEstDepth, int height, int width,int order,float dMin,float stepSz, int nDepth)
{
	if( !( Ir.rows == height ) || !( Ir.cols == width ) )
	{
		cout << " Input image is not of  same dimensions as heignt and width. Return;. " << endl; return;
	}

	bool DEBUG = 0;
	Mat qmat=Mat::zeros(height,width,CV_32FC1);
	float *dIr;
	float *dIm;
	float *dCost;
	float *dCounter;	
	float *dDepth;
	float *dMinm;
	float *dest;
	float *dP;
	float *dB;
	float *dConf;
	//int *dBorderFlag;
	int maxOrd=order;
	const int tPx = height*width;
	const int imgSz = Ir.cols*Ir.rows*sizeof(float);
													
	hipMalloc((void **)&dIr, imgSz);								
													
	SAFE_CALL(hipMemcpy(dIr,Ir.ptr(),imgSz,hipMemcpyHostToDevice),"CUDA Memcpy Host To Device Failed");//
	hipMalloc((void **)&dCost, nDepth*tPx*sizeof(float));						
	SAFE_CALL(hipMemset(dCost, 0, nDepth*tPx*sizeof(float)),"CUDA Memset Failed");			
	hipMalloc((void **)&dCounter, nDepth*tPx*sizeof(float));					
	hipMemset(dCounter, 0, nDepth*tPx*sizeof(int));
	hipMalloc((void **)&dDepth, imgSz);								
	hipMemset(dDepth, 0, tPx*sizeof(float));
	hipMalloc((void **)&dMinm, tPx*sizeof(float));							
	hipMemset(dMinm, std::numeric_limits<float>::infinity(), tPx*sizeof(float));//cout << " Value of dMinm :" << dMinm[0] <<endl;
													

	dim3 blSz(8,32,1);								
	dim3 thSz(nDepth/blSz.x, (tPx)/blSz.y, 1);
	dim3 blockSize(16, 16, 1);
	dim3 gridSize((Ir.cols)/blockSize.x, (Ir.rows)/blockSize.y, 1);	

	hipMalloc((void **)&dIm, imgSz);						
	hipMalloc((void **)&dConf, tPx*sizeof(float));					
	hipMalloc((void **)&dest, tPx*sizeof(float));					
	hipMalloc((void **)&dP, 3*3*sizeof(float));					
	hipMalloc((void **)&dB, 3*3*sizeof(float));					
											

	size_t pitch;
	float* d_texture; //Device texture
											
	hipMallocPitch((void**)&d_texture,&pitch, width * sizeof(float), height);
											
//--------------FOR LOOP............................
	for(int i = 0 ; i < Im.size() ; i++ )
		{
		Mat A = T[i]*Nv;//
		Mat P = K * R[i] * Ki;//
		Mat B = K * A * Ki;//
										
		hipMemcpy(dP, P.ptr(), 3*3*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dB, B.ptr(), 3*3*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy2D(d_texture, pitch, Im[i].ptr(), width * sizeof(float), width *sizeof(float), height, hipMemcpyHostToDevice);
		hipBindTexture2D(NULL, tex_Im, d_texture, tex_Im.channelDesc, width, height, pitch) ;
		//-------COstVolume Calculation------------------------------------------------------------------------------------------
		bool dLast = (i == (Im.size() -1));				
		costVolume<<<gridSize,blockSize>>>(dIr, dCost, dCounter, dDepth, dMinm, dLast, dConf, dest, dP, dB, height, width , maxOrd, nDepth);
		//-----------------------------------------------------------------------------------------------------------------------
													
		double minE, maxE;
		cv::minMaxLoc(confMat, &minE, &maxE);				
										
		hipUnbindTexture(tex_Im);					
		}
		cost_refinement<<<gridSize,blockSize>>>(dCost, dCounter, dDepth, dMinm, height, width , maxOrd, nDepth);
		DEBUG = 1;
											
		double minm, maxm;

		cv::Mat mDepth = cv::Mat::zeros(Ir.size(),IrEstDepth.type());
		hipMemcpy(mDepth.ptr(), dDepth, tPx*sizeof(float), hipMemcpyDeviceToHost);
		minMaxLoc(mDepth, &minm, &maxm);
											
		cv::Mat mMinm = cv::Mat::zeros(Ir.size(),IrEstDepth.type());
		hipMemcpy(mMinm.ptr(), dMinm, tPx*sizeof(float), hipMemcpyDeviceToHost);
		minMaxLoc(mMinm, &minm, &maxm);
		/*normalize(mDepth,mDepth,0,1,NORM_MINMAX);						if(DEBUG)cout<<"52 :minmax of mMinm : "<<minm<<" , "<<maxm<<endl;
		imshow("dDepth",mDepth);
		waitKey();*/
		IrEstDepth = mDepth.clone();
		cv::Mat mCost[nDepth];
		cv::Mat mCounter[nDepth];						
		hipFree(dIm);								
		hipFree(d_texture);
		hipFree(dConf);							
		hipFree(dest);								
		hipFree(dP);								
		hipFree(dB);								
		hipFree(dIr);//printf("Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));
		hipFree(dCost);
		hipFree(dCounter);
		hipFree(dMinm);
		hipFree(dDepth);	
}



